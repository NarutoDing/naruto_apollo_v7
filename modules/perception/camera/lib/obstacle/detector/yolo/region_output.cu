#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2018 The Apollo Authors. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the License);
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an AS IS BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/
#include <algorithm>
#include <functional>
#include <map>
#include <memory>
#include <utility>
#include <vector>

#include "boost/iterator/counting_iterator.hpp"
#include "thrust/functional.h"
#include "thrust/sort.h"

#include "modules/perception/base/object_types.h"
#include "modules/perception/camera/lib/obstacle/detector/yolo/object_maintainer.h"
#include "modules/perception/camera/lib/obstacle/detector/yolo/region_output.h"

namespace apollo {
namespace perception {
namespace camera {

__host__ __device__ float sigmoid_gpu(float x) { return 1.0 / (1.0 + exp(-x)); }

__host__ __device__ float bbox_size_gpu(const float *bbox,
                                        const bool normalized) {
  if (bbox[2] <= bbox[0] || bbox[3] <= bbox[1]) {
    // If bbox is invalid (e.g. xmax < xmin or ymax < ymin), return 0.
    return 0.f;  // NOLINT
  } else {
    const float width = bbox[2] - bbox[0];
    const float height = bbox[3] - bbox[1];
    if (normalized) {
      return width * height;
    } else {
      // If bbox is not within range [0, 1].
      return (width + 1) * (height + 1);
    }
  }
}

__host__ __device__ float jaccard_overlap_gpu(const float *bbox1,
                                              const float *bbox2) {
  if (bbox2[0] > bbox1[2] || bbox2[2] < bbox1[0] || bbox2[1] > bbox1[3] ||
      bbox2[3] < bbox1[1]) {
    return float(0.);  // NOLINT
  } else {
    const float inter_xmin = max(bbox1[0], bbox2[0]);
    const float inter_ymin = max(bbox1[1], bbox2[1]);
    const float inter_xmax = min(bbox1[2], bbox2[2]);
    const float inter_ymax = min(bbox1[3], bbox2[3]);

    const float inter_width = inter_xmax - inter_xmin;
    const float inter_height = inter_ymax - inter_ymin;
    const float inter_size = inter_width * inter_height;

    const float bbox1_size = bbox_size_gpu(bbox1, true);
    const float bbox2_size = bbox_size_gpu(bbox2, true);

    return inter_size / (bbox1_size + bbox2_size - inter_size);
  }
}

__global__ void get_object_kernel(
    int n, const float *loc_data, const float *obj_data, const float *cls_data,
    const float *ori_data, const float *dim_data, const float *lof_data,
    const float *lor_data, const float *area_id_data,
    const float *visible_ratio_data, const float *cut_off_ratio_data,
    const float *brvis_data, const float *brswt_data, const float *ltvis_data,
    const float *ltswt_data, const float *rtvis_data, const float *rtswt_data,
    const float *anchor_data, const float *expand_data, int width, int height,
    int num_anchors, int num_classes, float confidence_threshold,
    float light_vis_conf_threshold, float light_swt_conf_threshold,
    bool with_box3d, bool with_frbox, bool with_lights, bool with_ratios,
    bool multi_scale, int num_areas, float *res_box_data, float *res_cls_data,
    int res_cls_offset, int all_scales_num_candidates) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);
       i += blockDim.x * gridDim.x) {
    int box_block = kBoxBlockSize;

    int idx = i;
    int c = idx % num_anchors;
    idx = idx / num_anchors;
    int w = idx % width;
    idx = idx / width;
    int h = idx;
    int loc_index = (h * width + w) * num_anchors + c;
    int offset_loc = loc_index * 4;
    int offset_cls = loc_index * num_classes;
    float scale = obj_data[loc_index];
    float cx = (w + sigmoid_gpu(loc_data[offset_loc + 0])) / width;
    float cy = (h + sigmoid_gpu(loc_data[offset_loc + 1])) / height;
    float hw =
        exp(max(minExpPower, min(loc_data[offset_loc + 2], maxExpPower))) *
        anchor_data[2 * c] / width * 0.5;
    float hh =
        exp(max(minExpPower, min(loc_data[offset_loc + 3], maxExpPower))) *
        anchor_data[2 * c + 1] / height * 0.5;

    float max_prob = 0.f;
    int max_index = 0;
    for (int k = 0; k < num_classes; ++k) {
      float prob = cls_data[offset_cls + k] * scale;
      res_cls_data[k * all_scales_num_candidates + res_cls_offset + i] = prob;
      if (prob > max_prob) {
        max_prob = prob;
        max_index = k;
      }
    }
    res_cls_data[num_classes * all_scales_num_candidates + res_cls_offset + i] =
        max_prob;

    auto &&dst_ptr = res_box_data + i * box_block;
    hw += expand_data[max_index];
    dst_ptr[0] = cx - hw;
    dst_ptr[1] = cy - hh;
    dst_ptr[2] = cx + hw;
    dst_ptr[3] = cy + hh;

    if (with_box3d) {
      int offset_ori = loc_index * 2;
      dst_ptr[4] = atan2(ori_data[offset_ori + 1], ori_data[offset_ori]);

      int offset_dim = loc_index * 3;
      if (multi_scale) {
        offset_dim = loc_index * num_classes * 3 + max_index * 3;
      }
      dst_ptr[5] = dim_data[offset_dim + 0];
      dst_ptr[6] = dim_data[offset_dim + 1];
      dst_ptr[7] = dim_data[offset_dim + 2];
    }

    if (with_frbox) {
      {
        int offset_lof = loc_index * 4;
        auto &&src_ptr = lof_data + offset_lof;
        auto sb_x = src_ptr[0] * hw * 2 + cx;
        auto sb_y = src_ptr[1] * hh * 2 + cy;
        auto sb_hw = exp(src_ptr[2]) * hw;
        auto sb_hh = exp(src_ptr[3]) * hh;
        dst_ptr[8] = sb_x - sb_hw;
        dst_ptr[9] = sb_y - sb_hh;
        dst_ptr[10] = sb_x + sb_hw;
        dst_ptr[11] = sb_y + sb_hh;
      }

      {
        int offset_lor = loc_index * 4;
        auto &&src_ptr = lor_data + offset_lor;
        auto sb_x = src_ptr[0] * hw * 2 + cx;
        auto sb_y = src_ptr[1] * hh * 2 + cy;
        auto sb_hw = exp(src_ptr[2]) * hw;
        auto sb_hh = exp(src_ptr[3]) * hh;
        dst_ptr[12] = sb_x - sb_hw;
        dst_ptr[13] = sb_y - sb_hh;
        dst_ptr[14] = sb_x + sb_hw;
        dst_ptr[15] = sb_y + sb_hh;
      }
    }

    if (with_lights) {
      dst_ptr[16] = sigmoid_gpu(brvis_data[loc_index]);
      dst_ptr[17] = sigmoid_gpu(brswt_data[loc_index]);
      dst_ptr[18] = sigmoid_gpu(ltvis_data[loc_index]);
      dst_ptr[19] = sigmoid_gpu(ltswt_data[loc_index]);
      dst_ptr[20] = sigmoid_gpu(rtvis_data[loc_index]);
      dst_ptr[21] = sigmoid_gpu(rtswt_data[loc_index]);

      dst_ptr[16] = dst_ptr[16] > light_vis_conf_threshold ? dst_ptr[16] : 0;
      dst_ptr[18] = dst_ptr[18] > light_vis_conf_threshold ? dst_ptr[18] : 0;
      dst_ptr[20] = dst_ptr[20] > light_vis_conf_threshold ? dst_ptr[20] : 0;

      float swt_score = 0;
      swt_score = dst_ptr[16] * dst_ptr[17];
      dst_ptr[17] = swt_score > light_swt_conf_threshold ? swt_score : 0;

      swt_score = dst_ptr[18] * dst_ptr[19];
      dst_ptr[19] = swt_score > light_swt_conf_threshold ? swt_score : 0;

      swt_score = dst_ptr[20] * dst_ptr[21];
      dst_ptr[21] = swt_score > light_swt_conf_threshold ? swt_score : 0;
    }

    if (with_ratios) {
      // 0~3: cos2, left, visa, visb
      auto vis_pred = visible_ratio_data + loc_index * 4;
      auto vis_ptr = dst_ptr + 22;
      vis_ptr[0] = vis_ptr[1] = vis_ptr[2] = vis_ptr[3] = 0;
      const float hi_th = 0.75;
      const float lo_th = 1.f - hi_th;
      if (vis_pred[2] >= hi_th && vis_pred[3] >= hi_th) {  // 2 (1, 3)
        vis_ptr[0] = vis_pred[0];
        vis_ptr[1] = 1 - vis_pred[0];
      } else if (vis_pred[2] <= lo_th && vis_pred[3] >= hi_th) {  // 4 (3, 5)
        vis_ptr[2] = vis_pred[0];
        vis_ptr[1] = 1 - vis_pred[0];
      } else if (vis_pred[2] <= lo_th && vis_pred[3] <= lo_th) {  // 6 (5, 7)
        vis_ptr[2] = vis_pred[0];
        vis_ptr[3] = 1 - vis_pred[0];
      } else if (vis_pred[2] >= hi_th && vis_pred[3] <= lo_th) {  // 8 (7, 1)
        vis_ptr[0] = vis_pred[0];
        vis_ptr[3] = 1 - vis_pred[0];
      } else {
        vis_ptr[2] = vis_pred[0];
        if (vis_pred[1] > 0.5) {
          vis_ptr[1] = 1 - vis_pred[0];
        } else {
          vis_ptr[3] = 1 - vis_pred[0];
        }
      }

      int offset_cut = loc_index * 4;
      dst_ptr[26] = cut_off_ratio_data[offset_cut + 0];
      dst_ptr[27] = cut_off_ratio_data[offset_cut + 1];
      dst_ptr[28] = cut_off_ratio_data[offset_cut + 2];
      dst_ptr[29] = cut_off_ratio_data[offset_cut + 3];
    }

    if (num_areas > 0) {
      int offset_area_id = loc_index * num_areas;
      int max_area_id = 0;
      for (int area_id = 1; area_id < num_areas; ++area_id) {
        if (area_id_data[offset_area_id + area_id] >
            area_id_data[offset_area_id + max_area_id]) {
          max_area_id = area_id;
        }
      }
      dst_ptr[30] = max_area_id + 1;
      dst_ptr[31] = area_id_data[offset_area_id + max_area_id];
    }
  }
}

__global__ void get_rois_kernel(int num_bboxes, const float *loc_data,
                                const float *obj_data, const float *anchor_data,
                                int width, int height, int num_anchors,
                                float confidence_threshold, float *conf_data,
                                float *bbox_data) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_bboxes) {
    int offset_obj = idx;
    int offset_loc = idx * 4;

    int c = idx % num_anchors;
    idx /= num_anchors;
    int w = idx % width;
    idx /= width;
    int h = idx;

    float cx = (w + sigmoid_gpu(loc_data[offset_loc + 0])) / width;
    float cy = (h + sigmoid_gpu(loc_data[offset_loc + 1])) / height;
    float hw =
        exp(loc_data[offset_loc + 2]) * anchor_data[2 * c + 0] / width * 0.5;
    float hh =
        exp(loc_data[offset_loc + 3]) * anchor_data[2 * c + 1] / height * 0.5;

    const float &conf = obj_data[offset_obj];
    conf_data[offset_obj] = conf > confidence_threshold ? conf : 0;

    auto &&curr_bbox = bbox_data + offset_loc;
    curr_bbox[0] = cx - hw;
    curr_bbox[1] = cy - hh;
    curr_bbox[2] = cx + hw;
    curr_bbox[3] = cy + hh;
  }
}

__global__ void compute_overlapped_by_idx_kernel(
    const int nthreads, const float *bbox_data, const int bbox_step,
    const float overlap_threshold, const int *idx, const int num_idx,
    bool *overlapped_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < (nthreads);
       index += blockDim.x * gridDim.x) {
    const int j = index % num_idx;
    const int i = index / num_idx;
    if (i == j) {
      // Ignore same bbox.
      return;
    }
    // Compute overlap between i-th bbox and j-th bbox.
    const int start_loc_i = idx[i] * bbox_step;
    const int start_loc_j = idx[j] * bbox_step;
    const float overlap =
        jaccard_overlap_gpu(bbox_data + start_loc_i, bbox_data + start_loc_j);
    overlapped_data[index] = overlap > overlap_threshold;
  }
}

void compute_overlapped_by_idx_gpu(const int nthreads, const float *bbox_data,
                                   const int bbox_step,
                                   const float overlap_threshold,
                                   const int *idx, const int num_idx,
                                   bool *overlapped_data,
                                   const hipStream_t &stream) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  const int thread_size = 512;
  int block_size = (nthreads + thread_size - 1) / thread_size;
  compute_overlapped_by_idx_kernel<<<block_size, thread_size, 0, stream>>>(
      nthreads, bbox_data, bbox_step, overlap_threshold, idx, num_idx,
      overlapped_data);
}

void apply_nms_gpu(const float *bbox_data, const float *conf_data,
                   const std::vector<int> &origin_indices, const int bbox_step,
                   const float confidence_threshold, const int top_k,
                   const float nms_threshold, std::vector<int> *indices,
                   base::Blob<bool> *overlapped, base::Blob<int> *idx_sm,
                   const hipStream_t &stream) {
  // Keep part of detections whose scores are higher than confidence threshold.
  std::vector<int> idx;
  std::vector<float> confidences;
  for (auto i : origin_indices) {
    if (conf_data[i] > confidence_threshold) {
      idx.push_back(i);
      confidences.push_back(conf_data[i]);
    }
  }
  int num_remain = confidences.size();
  if (num_remain == 0) {
    return;
  }
  // Sort detections based on score.
  thrust::sort_by_key(&confidences[0], &confidences[0] + num_remain, &idx[0],
                      thrust::greater<float>());
  if (top_k > -1 && top_k < num_remain) {
    num_remain = top_k;
  }
  int *idx_data = (idx_sm->mutable_cpu_data());
  std::copy(idx.begin(), idx.begin() + num_remain, idx_data);

  overlapped->Reshape(std::vector<int>{num_remain, num_remain});
  bool *overlapped_data = (overlapped->mutable_gpu_data());

  compute_overlapped_by_idx_gpu(overlapped->count(), bbox_data, bbox_step,
                                nms_threshold, idx_sm->gpu_data(), num_remain,
                                overlapped_data, stream);

  // Do non-maximum suppression based on overlapped results.
  const bool *overlapped_results = (const bool *)overlapped->cpu_data();
  std::vector<int> selected_indices;

  apply_nms(overlapped_results, num_remain, &selected_indices);
  // Put back the selected information.
  for (size_t i = 0; i < selected_indices.size(); ++i) {
    indices->push_back(idx[selected_indices[i]]);
  }
}

void apply_nms(const bool *overlapped, const int num,
               std::vector<int> *indices) {
  std::vector<int> index_vec(boost::counting_iterator<int>(0),
                             boost::counting_iterator<int>(num));
  // Do nms.
  indices->clear();
  while (index_vec.size() != 0) {
    // Get the current highest score box.
    int best_idx = index_vec.front();
    indices->push_back(best_idx);
    // Erase the best box.
    index_vec.erase(index_vec.begin());

    for (std::vector<int>::iterator it = index_vec.begin();
         it != index_vec.end();) {
      int cur_idx = *it;

      // Remove it if necessary
      if (overlapped[best_idx * num + cur_idx]) {
        it = index_vec.erase(it);
      } else {
        ++it;
      }
    }
  }
}

const float *get_gpu_data(bool flag, const base::Blob<float> &blob) {
  return flag ? blob.gpu_data() : nullptr;
}

int get_objects_gpu(const YoloBlobs &yolo_blobs, const hipStream_t &stream,
    const std::vector<base::ObjectSubType> &types,
    const NMSParam &nms, const yolo::ModelParam &model_param,
    float light_vis_conf_threshold,
    float light_swt_conf_threshold,
    base::Blob<bool> *overlapped, base::Blob<int> *idx_sm,
    const std::map<base::ObjectSubType, std::vector<int>> &indices_cns,
    const std::map<base::ObjectSubType, std::vector<float>> &conf_scores_cns) {
  auto& indices = const_cast<std::map<base::ObjectSubType,
                      std::vector<int>>& >(indices_cns);
  auto& conf_scores = const_cast<std::map<base::ObjectSubType,
                      std::vector<float>>& >(conf_scores_cns);

  bool multi_scale = false;
  if (yolo_blobs.det2_obj_blob) {
    multi_scale = true;
  }
  int num_classes = types.size();
  int batch = yolo_blobs.det1_obj_blob->shape(0);
  int num_anchor = yolo_blobs.anchor_blob->shape(2);
  int num_anchor_per_scale = num_anchor;
  if (multi_scale) {
    num_anchor_per_scale /= numScales;
  }
  CHECK_EQ(batch, 1) << "batch size should be 1!";

  std::vector<int> height_vec, width_vec, num_candidates_vec;
  height_vec.push_back(yolo_blobs.det1_obj_blob->shape(1));
  width_vec.push_back(yolo_blobs.det1_obj_blob->shape(2));
  if (multi_scale) {
    height_vec.push_back(yolo_blobs.det2_obj_blob->shape(1));
    height_vec.push_back(yolo_blobs.det3_obj_blob->shape(1));
    width_vec.push_back(yolo_blobs.det2_obj_blob->shape(2));
    width_vec.push_back(yolo_blobs.det3_obj_blob->shape(2));
  }
  for (size_t i = 0; i < height_vec.size(); i++) {
    num_candidates_vec.push_back(height_vec[i] * width_vec[i] *
                                 num_anchor_per_scale);
  }

  const float *loc_data_vec[3] = {
      yolo_blobs.det1_loc_blob->gpu_data(),
      yolo_blobs.det2_loc_blob ? yolo_blobs.det2_loc_blob->gpu_data() : nullptr,
      yolo_blobs.det3_loc_blob ? yolo_blobs.det3_loc_blob->gpu_data()
                               : nullptr};
  const float *obj_data_vec[3] = {
      yolo_blobs.det1_obj_blob->gpu_data(),
      yolo_blobs.det2_obj_blob ? yolo_blobs.det2_obj_blob->gpu_data() : nullptr,
      yolo_blobs.det3_obj_blob ? yolo_blobs.det3_obj_blob->gpu_data()
                               : nullptr};
  const float *cls_data_vec[3] = {
      yolo_blobs.det1_cls_blob->gpu_data(),
      yolo_blobs.det2_cls_blob ? yolo_blobs.det2_cls_blob->gpu_data() : nullptr,
      yolo_blobs.det3_cls_blob ? yolo_blobs.det3_cls_blob->gpu_data()
                               : nullptr};
  const float *ori_data_vec[3] = {
      get_gpu_data(model_param.with_box3d(), *yolo_blobs.det1_ori_blob),
      multi_scale
          ? get_gpu_data(model_param.with_box3d(), *yolo_blobs.det2_ori_blob)
          : nullptr,
      multi_scale
          ? get_gpu_data(model_param.with_box3d(), *yolo_blobs.det3_ori_blob)
          : nullptr};
  const float *dim_data_vec[3] = {
      get_gpu_data(model_param.with_box3d(), *yolo_blobs.det1_dim_blob),
      multi_scale
          ? get_gpu_data(model_param.with_box3d(), *yolo_blobs.det2_dim_blob)
          : nullptr,
      multi_scale
          ? get_gpu_data(model_param.with_box3d(), *yolo_blobs.det3_dim_blob)
          : nullptr};

  // TODO[KaWai]: add 3 scale frbox data and light data.
  const float *lof_data =
      get_gpu_data(model_param.with_frbox(), *yolo_blobs.lof_blob);
  const float *lor_data =
      get_gpu_data(model_param.with_frbox(), *yolo_blobs.lor_blob);

  const float *area_id_data =
      get_gpu_data(model_param.num_areas() > 0, *yolo_blobs.area_id_blob);
  const float *visible_ratio_data =
      get_gpu_data(model_param.with_ratios(), *yolo_blobs.visible_ratio_blob);
  const float *cut_off_ratio_data =
      get_gpu_data(model_param.with_ratios(), *yolo_blobs.cut_off_ratio_blob);

  const auto &with_lights = model_param.with_lights();
  const float *brvis_data = get_gpu_data(with_lights, *yolo_blobs.brvis_blob);
  const float *brswt_data = get_gpu_data(with_lights, *yolo_blobs.brswt_blob);
  const float *ltvis_data = get_gpu_data(with_lights, *yolo_blobs.ltvis_blob);
  const float *ltswt_data = get_gpu_data(with_lights, *yolo_blobs.ltswt_blob);
  const float *rtvis_data = get_gpu_data(with_lights, *yolo_blobs.rtvis_blob);
  const float *rtswt_data = get_gpu_data(with_lights, *yolo_blobs.rtswt_blob);

  int all_scales_num_candidates = 0;
  for (size_t i = 0; i < num_candidates_vec.size(); i++) {
    all_scales_num_candidates += num_candidates_vec[i];
  }
  yolo_blobs.res_box_blob->Reshape(
      std::vector<int>{1, 1, all_scales_num_candidates, kBoxBlockSize});
  yolo_blobs.res_cls_blob->Reshape(
      std::vector<int>{1, 1, num_classes + 1, all_scales_num_candidates});

  float *res_box_data = yolo_blobs.res_box_blob->mutable_gpu_data();
  float *res_cls_data = yolo_blobs.res_cls_blob->mutable_gpu_data();
  const int thread_size = 512;
  // TODO[KaWai]: use different stream to process scales in parallel.
  int num_candidates_offset = 0;
  for (int i = 0; i < num_candidates_vec.size(); i++) {
    int block_size = (num_candidates_vec[i] + thread_size - 1) / thread_size;
    const float *loc_data = loc_data_vec[i];
    const float *obj_data = obj_data_vec[i];
    const float *cls_data = cls_data_vec[i];
    const float *ori_data = ori_data_vec[i];
    const float *dim_data = dim_data_vec[i];
    const float *anchor_data =
        yolo_blobs.anchor_blob->gpu_data() + num_anchor_per_scale * 2 * i;
    const float *expand_data = yolo_blobs.expand_blob->gpu_data();
    const int width = width_vec[i];
    const int height = height_vec[i];
    get_object_kernel<<<block_size, thread_size, 0, stream>>>(
        num_candidates_vec[i], loc_data, obj_data, cls_data, ori_data, dim_data,
        lof_data, lor_data, area_id_data, visible_ratio_data,
        cut_off_ratio_data, brvis_data, brswt_data, ltvis_data, ltswt_data,
        rtvis_data, rtswt_data, anchor_data, yolo_blobs.expand_blob->gpu_data(),
        width, height, num_anchor_per_scale, num_classes,
        model_param.confidence_threshold(), light_vis_conf_threshold,
        light_swt_conf_threshold, model_param.with_box3d(),
        model_param.with_frbox(), model_param.with_lights(),
        model_param.with_ratios(), multi_scale, model_param.num_areas(),
        res_box_data + num_candidates_offset * kBoxBlockSize, res_cls_data,
        num_candidates_offset, all_scales_num_candidates);
    hipStreamSynchronize(stream);
    num_candidates_offset += num_candidates_vec[i];
  }
  const float *cpu_cls_data = yolo_blobs.res_cls_blob->cpu_data();

  std::vector<int> all_indices(all_scales_num_candidates);
  std::iota(all_indices.begin(), all_indices.end(), 0);
  std::vector<int> rest_indices;

  int top_k = idx_sm->count();
  int num_kept = 0;
  // inter-cls NMS
  apply_nms_gpu(
      res_box_data, cpu_cls_data + num_classes * all_scales_num_candidates,
      all_indices, kBoxBlockSize, nms.inter_cls_conf_thresh, top_k,
      nms.inter_cls_nms_thresh, &rest_indices, overlapped, idx_sm, stream);
  for (int k = 0; k < num_classes; ++k) {
    apply_nms_gpu(res_box_data, cpu_cls_data + k * all_scales_num_candidates,
                  rest_indices, kBoxBlockSize,
                  model_param.confidence_threshold(), top_k, nms.threshold,
                  &(indices[types[k]]), overlapped, idx_sm, stream);
    num_kept += indices[types[k]].size();
    std::vector<float> conf_score(
        cpu_cls_data + k * all_scales_num_candidates,
        cpu_cls_data + (k + 1) * all_scales_num_candidates);
    conf_scores.insert(std::make_pair(types[k], conf_score));
    hipStreamSynchronize(stream);
  }

  return num_kept;
}

}  // namespace camera
}  // namespace perception
}  // namespace apollo
