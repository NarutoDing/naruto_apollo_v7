#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2018 The Apollo Authors. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the License);
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an AS IS BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/
#include <algorithm>
#include <functional>
#include <map>
#include <memory>
#include <utility>
#include <vector>
#include "boost/iterator/counting_iterator.hpp"
#include "thrust/functional.h"
#include "thrust/sort.h"

#include "modules/perception/base/object_types.h"
#include "modules/perception/camera/lib/obstacle/detector/yolov4/object_maintainer.h"
#include "modules/perception/camera/lib/obstacle/detector/yolov4/region_output.h"

namespace apollo {
namespace perception {
namespace camera {

__host__ __device__ float sigmoid_gpu(float x) { return 1.0 / (1.0 + exp(-x)); }

__host__ __device__ float bbox_size_gpu(const float *bbox,
                                        const bool normalized) {
  if (bbox[2] <= bbox[0] || bbox[3] <= bbox[1]) {
    // If bbox is invalid (e.g. xmax < xmin or ymax < ymin), return 0.
    return 0.f;  // NOLINT
  } else {
    const float width = bbox[2] - bbox[0];
    const float height = bbox[3] - bbox[1];
    if (normalized) {
      return width * height;
    } else {
      // If bbox is not within range [0, 1].
      return (width + 1) * (height + 1);
    }
  }
}

__host__ __device__ float jaccard_overlap_gpu(const float *bbox1,
                                              const float *bbox2) {
  if (bbox2[0] > bbox1[2] || bbox2[2] < bbox1[0] || bbox2[1] > bbox1[3] ||
      bbox2[3] < bbox1[1]) {
    return float(0.);  // NOLINT
  } else {
    const float inter_xmin = max(bbox1[0], bbox2[0]);
    const float inter_ymin = max(bbox1[1], bbox2[1]);
    const float inter_xmax = min(bbox1[2], bbox2[2]);
    const float inter_ymax = min(bbox1[3], bbox2[3]);

    const float inter_width = inter_xmax - inter_xmin;
    const float inter_height = inter_ymax - inter_ymin;
    const float inter_size = inter_width * inter_height;

    const float bbox1_size = bbox_size_gpu(bbox1, true);
    const float bbox2_size = bbox_size_gpu(bbox2, true);

    return inter_size / (bbox1_size + bbox2_size - inter_size);
  }
}

__global__ void get_object_kernel(
    int n, const float *loc_data, const float *obj_data, const float *cls_data,
    const float *ori_data, const float *dim_data, const float *lof_data,
    const float *lor_data, const float *area_id_data,
    const float *visible_ratio_data, const float *cut_off_ratio_data,
    const float *brvis_data, const float *brswt_data, const float *ltvis_data,
    const float *ltswt_data, const float *rtvis_data, const float *rtswt_data,
    const float *anchor_data, const float *expand_data, int width, int height,
    int num_anchors, int num_classes, float confidence_threshold,
    float light_vis_conf_threshold, float light_swt_conf_threshold,
    bool with_box3d, bool with_frbox, bool with_lights, bool with_ratios,
    bool multi_scale, int num_areas, float *res_box_data, float *res_cls_data,
    int res_cls_offset, int all_scales_num_candidates) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n);
       i += blockDim.x * gridDim.x) {
    int box_block = kBoxBlockSize;

    int idx = i;
    int c = idx % num_anchors;
    idx = idx / num_anchors;
    int w = idx % width;
    idx = idx / width;
    int h = idx;
    int loc_index = (h * width + w) * num_anchors + c;
    int offset_loc = loc_index * 4;
    int offset_cls = loc_index * num_classes;
    float scale = obj_data[loc_index];
    float cx = (w + sigmoid_gpu(loc_data[offset_loc + 0])) / width;
    float cy = (h + sigmoid_gpu(loc_data[offset_loc + 1])) / height;
    float hw =
        exp(max(minExpPower, min(loc_data[offset_loc + 2], maxExpPower))) *
        anchor_data[2 * c] / width * 0.5;
    float hh =
        exp(max(minExpPower, min(loc_data[offset_loc + 3], maxExpPower))) *
        anchor_data[2 * c + 1] / height * 0.5;

    float max_prob = 0.f;
    int max_index = 0;
    for (int k = 0; k < num_classes; ++k) {
      float prob = cls_data[offset_cls + k] * scale;
      res_cls_data[k * all_scales_num_candidates + res_cls_offset + i] = prob;
      if (prob > max_prob) {
        max_prob = prob;
        max_index = k;
      }
    }
    res_cls_data[num_classes * all_scales_num_candidates + res_cls_offset + i] =
        max_prob;

    auto &&dst_ptr = res_box_data + i * box_block;
    hw += expand_data[max_index];
    dst_ptr[0] = cx - hw;
    dst_ptr[1] = cy - hh;
    dst_ptr[2] = cx + hw;
    dst_ptr[3] = cy + hh;

    if (with_box3d) {
      int offset_ori = loc_index * 2;
      dst_ptr[4] = atan2(ori_data[offset_ori + 1], ori_data[offset_ori]);

      int offset_dim = loc_index * 3;
      if (multi_scale) {
        offset_dim = loc_index * num_classes * 3 + max_index * 3;
      }
      dst_ptr[5] = dim_data[offset_dim + 0];
      dst_ptr[6] = dim_data[offset_dim + 1];
      dst_ptr[7] = dim_data[offset_dim + 2];
    }

    if (with_frbox) {
      {
        int offset_lof = loc_index * 4;
        auto &&src_ptr = lof_data + offset_lof;
        auto sb_x = src_ptr[0] * hw * 2 + cx;
        auto sb_y = src_ptr[1] * hh * 2 + cy;
        auto sb_hw = exp(src_ptr[2]) * hw;
        auto sb_hh = exp(src_ptr[3]) * hh;
        dst_ptr[8] = sb_x - sb_hw;
        dst_ptr[9] = sb_y - sb_hh;
        dst_ptr[10] = sb_x + sb_hw;
        dst_ptr[11] = sb_y + sb_hh;
      }

      {
        int offset_lor = loc_index * 4;
        auto &&src_ptr = lor_data + offset_lor;
        auto sb_x = src_ptr[0] * hw * 2 + cx;
        auto sb_y = src_ptr[1] * hh * 2 + cy;
        auto sb_hw = exp(src_ptr[2]) * hw;
        auto sb_hh = exp(src_ptr[3]) * hh;
        dst_ptr[12] = sb_x - sb_hw;
        dst_ptr[13] = sb_y - sb_hh;
        dst_ptr[14] = sb_x + sb_hw;
        dst_ptr[15] = sb_y + sb_hh;
      }
    }

    if (with_lights) {
      dst_ptr[16] = sigmoid_gpu(brvis_data[loc_index]);
      dst_ptr[17] = sigmoid_gpu(brswt_data[loc_index]);
      dst_ptr[18] = sigmoid_gpu(ltvis_data[loc_index]);
      dst_ptr[19] = sigmoid_gpu(ltswt_data[loc_index]);
      dst_ptr[20] = sigmoid_gpu(rtvis_data[loc_index]);
      dst_ptr[21] = sigmoid_gpu(rtswt_data[loc_index]);

      dst_ptr[16] = dst_ptr[16] > light_vis_conf_threshold ? dst_ptr[16] : 0;
      dst_ptr[18] = dst_ptr[18] > light_vis_conf_threshold ? dst_ptr[18] : 0;
      dst_ptr[20] = dst_ptr[20] > light_vis_conf_threshold ? dst_ptr[20] : 0;

      float swt_score = 0;
      swt_score = dst_ptr[16] * dst_ptr[17];
      dst_ptr[17] = swt_score > light_swt_conf_threshold ? swt_score : 0;

      swt_score = dst_ptr[18] * dst_ptr[19];
      dst_ptr[19] = swt_score > light_swt_conf_threshold ? swt_score : 0;

      swt_score = dst_ptr[20] * dst_ptr[21];
      dst_ptr[21] = swt_score > light_swt_conf_threshold ? swt_score : 0;
    }

    if (with_ratios) {
      // 0~3: cos2, left, visa, visb
      auto vis_pred = visible_ratio_data + loc_index * 4;
      auto vis_ptr = dst_ptr + 22;
      vis_ptr[0] = vis_ptr[1] = vis_ptr[2] = vis_ptr[3] = 0;
      const float hi_th = 0.75;
      const float lo_th = 1.f - hi_th;
      if (vis_pred[2] >= hi_th && vis_pred[3] >= hi_th) {  // 2 (1, 3)
        vis_ptr[0] = vis_pred[0];
        vis_ptr[1] = 1 - vis_pred[0];
      } else if (vis_pred[2] <= lo_th && vis_pred[3] >= hi_th) {  // 4 (3, 5)
        vis_ptr[2] = vis_pred[0];
        vis_ptr[1] = 1 - vis_pred[0];
      } else if (vis_pred[2] <= lo_th && vis_pred[3] <= lo_th) {  // 6 (5, 7)
        vis_ptr[2] = vis_pred[0];
        vis_ptr[3] = 1 - vis_pred[0];
      } else if (vis_pred[2] >= hi_th && vis_pred[3] <= lo_th) {  // 8 (7, 1)
        vis_ptr[0] = vis_pred[0];
        vis_ptr[3] = 1 - vis_pred[0];
      } else {
        vis_ptr[2] = vis_pred[0];
        if (vis_pred[1] > 0.5) {
          vis_ptr[1] = 1 - vis_pred[0];
        } else {
          vis_ptr[3] = 1 - vis_pred[0];
        }
      }

      int offset_cut = loc_index * 4;
      dst_ptr[26] = cut_off_ratio_data[offset_cut + 0];
      dst_ptr[27] = cut_off_ratio_data[offset_cut + 1];
      dst_ptr[28] = cut_off_ratio_data[offset_cut + 2];
      dst_ptr[29] = cut_off_ratio_data[offset_cut + 3];
    }

    if (num_areas > 0) {
      int offset_area_id = loc_index * num_areas;
      int max_area_id = 0;
      for (int area_id = 1; area_id < num_areas; ++area_id) {
        if (area_id_data[offset_area_id + area_id] >
            area_id_data[offset_area_id + max_area_id]) {
          max_area_id = area_id;
        }
      }
      dst_ptr[30] = max_area_id + 1;
      dst_ptr[31] = area_id_data[offset_area_id + max_area_id];
    }
  }
}

__global__ void get_rois_kernel(int num_bboxes, const float *loc_data,
                                const float *obj_data, const float *anchor_data,
                                int width, int height, int num_anchors,
                                float confidence_threshold, float *conf_data,
                                float *bbox_data) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_bboxes) {
    int offset_obj = idx;
    int offset_loc = idx * 4;

    int c = idx % num_anchors;
    idx /= num_anchors;
    int w = idx % width;
    idx /= width;
    int h = idx;

    float cx = (w + sigmoid_gpu(loc_data[offset_loc + 0])) / width;
    float cy = (h + sigmoid_gpu(loc_data[offset_loc + 1])) / height;
    float hw =
        exp(loc_data[offset_loc + 2]) * anchor_data[2 * c + 0] / width * 0.5;
    float hh =
        exp(loc_data[offset_loc + 3]) * anchor_data[2 * c + 1] / height * 0.5;

    const float &conf = obj_data[offset_obj];
    conf_data[offset_obj] = conf > confidence_threshold ? conf : 0;

    auto &&curr_bbox = bbox_data + offset_loc;
    curr_bbox[0] = cx - hw;
    curr_bbox[1] = cy - hh;
    curr_bbox[2] = cx + hw;
    curr_bbox[3] = cy + hh;
  }
}

__global__ void compute_overlapped_by_idx_kernel(
    const int nthreads, const float *bbox_data, const int bbox_step,
    const float overlap_threshold, const int *idx, const int num_idx,
    bool *overlapped_data) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < (nthreads);
       index += blockDim.x * gridDim.x) {
    const int j = index % num_idx;
    const int i = index / num_idx;
    if (i == j) {
      // Ignore same bbox.
      return;
    }
    // Compute overlap between i-th bbox and j-th bbox.
    const int start_loc_i = idx[i] * bbox_step;
    const int start_loc_j = idx[j] * bbox_step;
    const float overlap =
        jaccard_overlap_gpu(bbox_data + start_loc_i, bbox_data + start_loc_j);
    overlapped_data[index] = overlap > overlap_threshold;
  }
}

void compute_overlapped_by_idx_gpu(const int nthreads, const float *bbox_data,
                                   const int bbox_step,
                                   const float overlap_threshold,
                                   const int *idx, const int num_idx,
                                   bool *overlapped_data,
                                   const hipStream_t &stream) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  const int thread_size = 512;
  int block_size = (nthreads + thread_size - 1) / thread_size;
  compute_overlapped_by_idx_kernel<<<block_size, thread_size, 0, stream>>>(
      nthreads, bbox_data, bbox_step, overlap_threshold, idx, num_idx,
      overlapped_data);
}

void apply_nms_gpu(const float *bbox_data, const float *conf_data,
                   const std::vector<int> &origin_indices, const int bbox_step,
                   const float confidence_threshold, const int top_k,
                   const float nms_threshold, std::vector<int> *indices,
                   base::Blob<bool> *overlapped, base::Blob<int> *idx_sm,
                   const hipStream_t &stream) {
  // Keep part of detections whose scores are higher than confidence threshold.
  std::vector<int> idx;
  std::vector<float> confidences;
  for (auto i : origin_indices) {
    if (conf_data[i] > confidence_threshold) {
      idx.push_back(i);
      confidences.push_back(conf_data[i]);
    }
  }
  int num_remain = confidences.size();
  if (num_remain == 0) {
    return;
  }
  // Sort detections based on score.
  thrust::sort_by_key(&confidences[0], &confidences[0] + num_remain, &idx[0],
                      thrust::greater<float>());
  if (top_k > -1 && top_k < num_remain) {
    num_remain = top_k;
  }
  int *idx_data = (idx_sm->mutable_cpu_data());
  std::copy(idx.begin(), idx.begin() + num_remain, idx_data);

  overlapped->Reshape(std::vector<int>{num_remain, num_remain});
  bool *overlapped_data = (overlapped->mutable_gpu_data());

  compute_overlapped_by_idx_gpu(overlapped->count(), bbox_data, bbox_step,
                                nms_threshold, idx_sm->gpu_data(), num_remain,
                                overlapped_data, stream);

  // Do non-maximum suppression based on overlapped results.
  const bool *overlapped_results = (const bool *)overlapped->cpu_data();
  std::vector<int> selected_indices;

  apply_nms(overlapped_results, num_remain, &selected_indices);
  // Put back the selected information.
  for (size_t i = 0; i < selected_indices.size(); ++i) {
    indices->push_back(idx[selected_indices[i]]);
  }
}

void apply_nms(const bool *overlapped, const int num,
               std::vector<int> *indices) {
  std::vector<int> index_vec(boost::counting_iterator<int>(0),
                             boost::counting_iterator<int>(num));
  // Do nms.
  indices->clear();
  while (index_vec.size() != 0) {
    // Get the current highest score box.
    int best_idx = index_vec.front();
    indices->push_back(best_idx);
    // Erase the best box.
    index_vec.erase(index_vec.begin());

    for (std::vector<int>::iterator it = index_vec.begin();
         it != index_vec.end();) {
      int cur_idx = *it;

      // Remove it if necessary
      if (overlapped[best_idx * num + cur_idx]) {
        it = index_vec.erase(it);
      } else {
        ++it;
      }
    }
  }
}

const float *get_gpu_data(bool flag, const base::Blob<float> &blob) {
  return flag ? blob.gpu_data() : nullptr;
}

void get_objects_gpu(const YoloBlobs &yolo_blobs, const hipStream_t &stream,
                     const std::vector<base::ObjectSubType> &types,
                     const NMSParam &nms, const yolo::ModelParam &model_param,
                     float light_vis_conf_threshold,
                     float light_swt_conf_threshold,
                     base::Blob<bool> *overlapped, base::Blob<int> *idx_sm,
                     std::vector<base::ObjectPtr> *objects) {
  bool multi_scale = false;
  if (yolo_blobs.det2_obj_blob) {
    multi_scale = true;
  }
  int num_classes = types.size();
  int batch = yolo_blobs.det1_obj_blob->shape(0);
  int num_anchor = yolo_blobs.anchor_blob->shape(2);
  int num_anchor_per_scale = num_anchor;
  if (multi_scale) {
    num_anchor_per_scale /= numScales;
  }
  CHECK_EQ(batch, 1) << "batch size should be 1!";

  std::vector<int> height_vec, width_vec, num_candidates_vec;
  height_vec.push_back(yolo_blobs.det1_obj_blob->shape(1));
  width_vec.push_back(yolo_blobs.det1_obj_blob->shape(2));
  if (multi_scale) {
    height_vec.push_back(yolo_blobs.det2_obj_blob->shape(1));
    height_vec.push_back(yolo_blobs.det3_obj_blob->shape(1));
    width_vec.push_back(yolo_blobs.det2_obj_blob->shape(2));
    width_vec.push_back(yolo_blobs.det3_obj_blob->shape(2));
  }
  for (size_t i = 0; i < height_vec.size(); i++) {
    num_candidates_vec.push_back(height_vec[i] * width_vec[i] *
                                 num_anchor_per_scale);
  }

  const float *loc_data_vec[3] = {
      yolo_blobs.det1_loc_blob->gpu_data(),
      yolo_blobs.det2_loc_blob ? yolo_blobs.det2_loc_blob->gpu_data() : nullptr,
      yolo_blobs.det3_loc_blob ? yolo_blobs.det3_loc_blob->gpu_data()
                               : nullptr};
  const float *obj_data_vec[3] = {
      yolo_blobs.det1_obj_blob->gpu_data(),
      yolo_blobs.det2_obj_blob ? yolo_blobs.det2_obj_blob->gpu_data() : nullptr,
      yolo_blobs.det3_obj_blob ? yolo_blobs.det3_obj_blob->gpu_data()
                               : nullptr};
  const float *cls_data_vec[3] = {
      yolo_blobs.det1_cls_blob->gpu_data(),
      yolo_blobs.det2_cls_blob ? yolo_blobs.det2_cls_blob->gpu_data() : nullptr,
      yolo_blobs.det3_cls_blob ? yolo_blobs.det3_cls_blob->gpu_data()
                               : nullptr};
  const float *ori_data_vec[3] = {
      get_gpu_data(model_param.with_box3d(), *yolo_blobs.det1_ori_blob),
      multi_scale
          ? get_gpu_data(model_param.with_box3d(), *yolo_blobs.det2_ori_blob)
          : nullptr,
      multi_scale
          ? get_gpu_data(model_param.with_box3d(), *yolo_blobs.det3_ori_blob)
          : nullptr};
  const float *dim_data_vec[3] = {
      get_gpu_data(model_param.with_box3d(), *yolo_blobs.det1_dim_blob),
      multi_scale
          ? get_gpu_data(model_param.with_box3d(), *yolo_blobs.det2_dim_blob)
          : nullptr,
      multi_scale
          ? get_gpu_data(model_param.with_box3d(), *yolo_blobs.det3_dim_blob)
          : nullptr};

  // TODO[KaWai]: add 3 scale frbox data and light data.
  const float *lof_data =
      get_gpu_data(model_param.with_frbox(), *yolo_blobs.lof_blob);
  const float *lor_data =
      get_gpu_data(model_param.with_frbox(), *yolo_blobs.lor_blob);

  const float *area_id_data =
      get_gpu_data(model_param.num_areas() > 0, *yolo_blobs.area_id_blob);
  const float *visible_ratio_data =
      get_gpu_data(model_param.with_ratios(), *yolo_blobs.visible_ratio_blob);
  const float *cut_off_ratio_data =
      get_gpu_data(model_param.with_ratios(), *yolo_blobs.cut_off_ratio_blob);

  const auto &with_lights = model_param.with_lights();
  const float *brvis_data = get_gpu_data(with_lights, *yolo_blobs.brvis_blob);
  const float *brswt_data = get_gpu_data(with_lights, *yolo_blobs.brswt_blob);
  const float *ltvis_data = get_gpu_data(with_lights, *yolo_blobs.ltvis_blob);
  const float *ltswt_data = get_gpu_data(with_lights, *yolo_blobs.ltswt_blob);
  const float *rtvis_data = get_gpu_data(with_lights, *yolo_blobs.rtvis_blob);
  const float *rtswt_data = get_gpu_data(with_lights, *yolo_blobs.rtswt_blob);

  int all_scales_num_candidates = 0;
  for (size_t i = 0; i < num_candidates_vec.size(); i++) {
    all_scales_num_candidates += num_candidates_vec[i];
  }
  yolo_blobs.res_box_blob->Reshape(
      std::vector<int>{1, 1, all_scales_num_candidates, kBoxBlockSize});
  yolo_blobs.res_cls_blob->Reshape(
      std::vector<int>{1, 1, num_classes + 1, all_scales_num_candidates});

  float *res_box_data = yolo_blobs.res_box_blob->mutable_gpu_data();
  float *res_cls_data = yolo_blobs.res_cls_blob->mutable_gpu_data();
  const int thread_size = 512;
  // TODO[KaWai]: use different stream to process scales in parallel.
  int num_candidates_offset = 0;
  for (int i = 0; i < num_candidates_vec.size(); i++) {
    int block_size = (num_candidates_vec[i] + thread_size - 1) / thread_size;
    const float *loc_data = loc_data_vec[i];
    const float *obj_data = obj_data_vec[i];
    const float *cls_data = cls_data_vec[i];
    const float *ori_data = ori_data_vec[i];
    const float *dim_data = dim_data_vec[i];
    const float *anchor_data =
        yolo_blobs.anchor_blob->gpu_data() + num_anchor_per_scale * 2 * i;
    const float *expand_data = yolo_blobs.expand_blob->gpu_data();
    const int width = width_vec[i];
    const int height = height_vec[i];
    get_object_kernel<<<block_size, thread_size, 0, stream>>>(
        num_candidates_vec[i], loc_data, obj_data, cls_data, ori_data, dim_data,
        lof_data, lor_data, area_id_data, visible_ratio_data,
        cut_off_ratio_data, brvis_data, brswt_data, ltvis_data, ltswt_data,
        rtvis_data, rtswt_data, anchor_data, yolo_blobs.expand_blob->gpu_data(),
        width, height, num_anchor_per_scale, num_classes,
        model_param.confidence_threshold(), light_vis_conf_threshold,
        light_swt_conf_threshold, model_param.with_box3d(),
        model_param.with_frbox(), model_param.with_lights(),
        model_param.with_ratios(), multi_scale, model_param.num_areas(),
        res_box_data + num_candidates_offset * kBoxBlockSize, res_cls_data,
        num_candidates_offset, all_scales_num_candidates);
    hipStreamSynchronize(stream);
    num_candidates_offset += num_candidates_vec[i];
  }
  const float *cpu_cls_data = yolo_blobs.res_cls_blob->cpu_data();

  std::vector<int> all_indices(all_scales_num_candidates);
  std::iota(all_indices.begin(), all_indices.end(), 0);
  std::vector<int> rest_indices;

  std::map<base::ObjectSubType, std::vector<int>> indices;
  std::map<base::ObjectSubType, std::vector<float>> conf_scores;

  int top_k = idx_sm->count();
  int num_kept = 0;
  // inter-cls NMS
  apply_nms_gpu(
      res_box_data, cpu_cls_data + num_classes * all_scales_num_candidates,
      all_indices, kBoxBlockSize, nms.inter_cls_conf_thresh, top_k,
      nms.inter_cls_nms_thresh, &rest_indices, overlapped, idx_sm, stream);
  for (int k = 0; k < num_classes; ++k) {
    apply_nms_gpu(res_box_data, cpu_cls_data + k * all_scales_num_candidates,
                  rest_indices, kBoxBlockSize,
                  model_param.confidence_threshold(), top_k, nms.threshold,
                  &(indices[types[k]]), overlapped, idx_sm, stream);
    num_kept += indices[types[k]].size();
    std::vector<float> conf_score(
        cpu_cls_data + k * all_scales_num_candidates,
        cpu_cls_data + (k + 1) * all_scales_num_candidates);
    conf_scores.insert(std::make_pair(types[k], conf_score));
    hipStreamSynchronize(stream);
  }

  objects->clear();

  if (num_kept == 0) {
    return;
  }

  objects->reserve(num_kept);
  const float *cpu_box_data = yolo_blobs.res_box_blob->cpu_data();

  ObjectMaintainer maintainer;
  for (auto it = indices.begin(); it != indices.end(); ++it) {
    base::ObjectSubType label = it->first;
    if (conf_scores.find(label) == conf_scores.end()) {
      // Something bad happened if there are no predictions for current label.
      continue;
    }
    const std::vector<float> &scores = conf_scores.find(label)->second;
    std::vector<int> &indice = it->second;
    for (size_t j = 0; j < indice.size(); ++j) {
      int idx = indice[j];
      const float *bbox = cpu_box_data + idx * kBoxBlockSize;
      if (scores[idx] < model_param.confidence_threshold()) {
        continue;
      }

      base::ObjectPtr obj = nullptr;
      obj.reset(new base::Object);
      obj->type = base::kSubType2TypeMap.at(label);
      obj->sub_type = label;
      obj->type_probs.assign(
          static_cast<int>(base::ObjectType::MAX_OBJECT_TYPE), 0);
      obj->sub_type_probs.assign(
          static_cast<int>(base::ObjectSubType::MAX_OBJECT_TYPE), 0);
      float total = 1e-5;
      for (int k = 0; k < num_classes; ++k) {
        auto &vis_type_k = types[k];
        auto &obj_type_k = base::kSubType2TypeMap.at(vis_type_k);
        auto &conf_score = conf_scores[vis_type_k][idx];
        obj->type_probs[static_cast<int>(obj_type_k)] += conf_score;
        obj->sub_type_probs[static_cast<int>(vis_type_k)] = conf_score;
        total += conf_score;
      }
      obj->confidence = obj->type_probs[static_cast<int>(obj->type)];
      for (int k = 0; k < obj->type_probs.size(); ++k) {
        obj->type_probs[k] /= total;
      }
      fill_base(obj, bbox);
      fill_bbox3d(model_param.with_box3d(), obj, bbox + 4);
      fill_frbox(model_param.with_frbox(), obj, bbox + 8);
      fill_lights(model_param.with_lights(), obj, bbox + 16);
      fill_ratios(model_param.with_ratios(), obj, bbox + 22);
      fill_area_id(model_param.num_areas() > 0, obj, bbox + 30);

      if (maintainer.Add(idx, obj)) {
        objects->push_back(obj);
      }
    }
  }
}

void get_intersect_bbox(const NormalizedBBox &bbox1,
                        const NormalizedBBox &bbox2,
                        NormalizedBBox *intersect_bbox) {
  if (bbox2.xmin > bbox1.xmax || bbox2.xmax < bbox1.xmin ||
      bbox2.ymin > bbox1.ymax || bbox2.ymax < bbox1.ymin) {
    // Return [0, 0, 0, 0] if there is no intersection.
    intersect_bbox->xmin = 0;
    intersect_bbox->ymin = 0;
    intersect_bbox->xmax = 0;
    intersect_bbox->ymax = 0;
  } else {
    intersect_bbox->xmin = std::max(bbox1.xmin, bbox2.xmin);
    intersect_bbox->ymin = std::max(bbox1.ymin, bbox2.ymin);
    intersect_bbox->xmax = std::min(bbox1.xmax, bbox2.xmax);
    intersect_bbox->ymax = std::min(bbox1.ymax, bbox2.ymax);
  }
}

float get_bbox_size(const NormalizedBBox &bbox) {
  if (bbox.xmax < bbox.xmin || bbox.ymax < bbox.ymin) {
    // If bbox is invalid (e.g. xmax < xmin or ymax < ymin), return 0.
    return 0;
  } else {
    if (bbox.size >= 0) {
      return bbox.size;
    } else {
      float width = bbox.xmax - bbox.xmin;
      float height = bbox.ymax - bbox.ymin;
      return width * height;
    }
  }
}

float get_jaccard_overlap(const NormalizedBBox &bbox1,
                          const NormalizedBBox &bbox2) {
  NormalizedBBox intersect_bbox;
  get_intersect_bbox(bbox1, bbox2, &intersect_bbox);
  float intersect_width = 0.f;
  float intersect_height = 0.f;
  intersect_width = intersect_bbox.xmax - intersect_bbox.xmin;
  intersect_height = intersect_bbox.ymax - intersect_bbox.ymin;

  if (intersect_width > 0 && intersect_height > 0) {
    float intersect_size = intersect_width * intersect_height;
    float bbox1_size = get_bbox_size(bbox1);
    float bbox2_size = get_bbox_size(bbox2);
    return intersect_size / (bbox1_size + bbox2_size - intersect_size);
  } else {
    return 0.;
  }
}

void get_max_score_index(const std::vector<float> &scores,
                         const float threshold, const int top_k,
                         std::vector<std::pair<float, int>> *score_index_vec) {
  // Generate index score pairs.
  for (size_t i = 0; i < scores.size(); ++i) {
    if (scores[i] > threshold) {
      score_index_vec->push_back(std::make_pair(scores[i], i));
    }
  }

  // Sort the score pair according to the scores in descending order
  std::stable_sort(score_index_vec->begin(), score_index_vec->end(),
                   sort_score_pair_descend<int>);

  // Keep top_k scores if needed.
  if (top_k > -1 && top_k < static_cast<int>(score_index_vec->size())) {
    score_index_vec->resize(top_k);
  }
}

void apply_softnms_fast(const std::vector<NormalizedBBox> &bboxes,
                        std::vector<float> *scores, const float score_threshold,
                        const float nms_threshold, const int top_k,
                        std::vector<int> *indices, bool is_linear,
                        const float sigma) {
  // Sanity check.
  CHECK_EQ(bboxes.size(), scores->size())
      << "bboxes and scores have different size.";

  // Get top_k scores (with corresponding indices).
  std::vector<std::pair<float, int>> score_index_vec;
  get_max_score_index(*scores, score_threshold, top_k, &score_index_vec);

  // Do nms.
  indices->clear();
  while (score_index_vec.size() != 0) {
    auto best_it =
        max_element(std::begin(score_index_vec), std::end(score_index_vec));
    const int best_idx = (*best_it).second;
    score_index_vec.erase(best_it);
    const NormalizedBBox &best_bbox = bboxes[best_idx];
    indices->push_back(best_idx);
    for (std::vector<std::pair<float, int>>::iterator it =
             score_index_vec.begin();
         it != score_index_vec.end();) {
      int cur_idx = it->second;
      const NormalizedBBox &cur_bbox = bboxes[cur_idx];

      float cur_overlap = 0.;
      cur_overlap = get_jaccard_overlap(best_bbox, cur_bbox);
      if (is_linear) {
        (*scores)[cur_idx] *= (1.0 - cur_overlap);
      } else {
        (*scores)[cur_idx] *= exp(-1.0 * pow(cur_overlap, 2) / sigma);
      }
      ++it;
    }
  }
}

void apply_boxvoting_fast(std::vector<NormalizedBBox> *bboxes,
                          std::vector<float> *scores,
                          const float conf_threshold, const float nms_threshold,
                          const float sigma, std::vector<int> *indices) {
  if (bboxes->size() == 0) {
    return;
  }
  indices->clear();
  for (size_t i = 0; i < bboxes->size(); ++i) {
    (*bboxes)[i].mask = false;
    if ((*scores)[i] > conf_threshold) {
      indices->push_back(i);
    }
  }
  for (size_t count = 0; count < indices->size(); ++count) {
    int max_box_idx = 0;

    for (size_t i = 1; i < indices->size(); ++i) {
      int idx = indices->at(i);
      if ((*bboxes)[idx].mask) {
        continue;
      }
      if ((*scores)[idx] > (*scores)[max_box_idx]) {
        max_box_idx = idx;
      }
    }

    NormalizedBBox &best_bbox = (*bboxes)[max_box_idx];
    best_bbox.score = (*scores)[max_box_idx];
    best_bbox.mask = true;
    float s_vt = (*scores)[max_box_idx];
    float x1_vt = best_bbox.xmin * s_vt;
    float x2_vt = best_bbox.xmax * s_vt;
    float y1_vt = best_bbox.ymin * s_vt;
    float y2_vt = best_bbox.ymax * s_vt;
    for (size_t i = 0; i < indices->size(); ++i) {
      int sub_it = indices->at(i);
      if ((*bboxes)[sub_it].mask) {
        continue;
      }
      float cur_overlap = 0.;
      cur_overlap = get_jaccard_overlap(best_bbox, (*bboxes)[sub_it]);
      if (sigma == 0) {
        (*bboxes)[sub_it].mask = true;
      } else {
        (*scores)[sub_it] *= exp(-1.0 * pow(cur_overlap, 2) / sigma);
      }
      (*bboxes)[sub_it].score = (*scores)[sub_it];

      // Remove it if necessary
      if (cur_overlap > nms_threshold) {
        float s_vt_cur = (*bboxes)[sub_it].score;
        s_vt += s_vt_cur;
        x1_vt += (*bboxes)[sub_it].xmin * s_vt_cur;
        x2_vt += (*bboxes)[sub_it].xmax * s_vt_cur;
        y1_vt += (*bboxes)[sub_it].ymin * s_vt_cur;
        y2_vt += (*bboxes)[sub_it].ymax * s_vt_cur;
      }
    }
    if (s_vt > 0.0001) {
      (*bboxes)[max_box_idx].xmin = x1_vt / s_vt;
      (*bboxes)[max_box_idx].xmax = x2_vt / s_vt;
      (*bboxes)[max_box_idx].ymin = y1_vt / s_vt;
      (*bboxes)[max_box_idx].ymax = y2_vt / s_vt;
    }
  }
}

void apply_nms_fast(const std::vector<NormalizedBBox> &bboxes,
                    const std::vector<float> &scores,
                    const float score_threshold, const float nms_threshold,
                    const float eta, const int top_k,
                    std::vector<int> *indices) {
  // Sanity check.
  CHECK_EQ(bboxes.size(), scores.size())
      << "bboxes and scores have different size.";

  // Get top_k scores (with corresponding indices).
  std::vector<std::pair<float, int>> score_index_vec;
  get_max_score_index(scores, score_threshold, top_k, &score_index_vec);

  // Do nms.
  float adaptive_threshold = nms_threshold;
  indices->clear();
  while (score_index_vec.size() != 0) {
    const int idx = score_index_vec.front().second;
    bool keep = true;
    for (size_t k = 0; k < indices->size(); ++k) {
      if (keep) {
        const int kept_idx = (*indices)[k];
        float overlap = get_jaccard_overlap(bboxes[idx], bboxes[kept_idx]);
        keep = overlap <= adaptive_threshold;
      } else {
        break;
      }
    }
    if (keep) {
      indices->push_back(idx);
    }
    score_index_vec.erase(score_index_vec.begin());
    if (keep && eta < 1 && adaptive_threshold > 0.5) {
      adaptive_threshold *= eta;
    }
  }
}

void filter_bbox(const MinDims &min_dims,
                 std::vector<base::ObjectPtr> *objects) {
  size_t valid_obj_idx = 0;
  size_t total_obj_idx = 0;
  while (total_obj_idx < objects->size()) {
    const auto &obj = (*objects)[total_obj_idx];
    if ((obj->camera_supplement.box.ymax - obj->camera_supplement.box.ymin) >=
            min_dims.min_2d_height &&
        (min_dims.min_3d_height <= 0 ||
         obj->size[2] >= min_dims.min_3d_height) &&
        (min_dims.min_3d_width <= 0 || obj->size[1] >= min_dims.min_3d_width) &&
        (min_dims.min_3d_length <= 0 ||
         obj->size[0] >= min_dims.min_3d_length)) {
      (*objects)[valid_obj_idx] = (*objects)[total_obj_idx];
      ++valid_obj_idx;
    }
    ++total_obj_idx;
  }
  objects->resize(valid_obj_idx);
}
void recover_bbox(int roi_w, int roi_h, int offset_y,
                  std::vector<base::ObjectPtr> *objects) {
  for (auto &obj : *objects) {
    float xmin = obj->camera_supplement.box.xmin;
    float ymin = obj->camera_supplement.box.ymin;
    float xmax = obj->camera_supplement.box.xmax;
    float ymax = obj->camera_supplement.box.ymax;
    int x = xmin * roi_w;
    int w = (xmax - xmin) * roi_w;
    int y = ymin * roi_h + offset_y;
    int h = (ymax - ymin) * roi_h;
    base::RectF rect_det(x, y, w, h);
    base::RectF rect_img(0, 0, roi_w, roi_h + offset_y);
    base::RectF rect = rect_det & rect_img;
    obj->camera_supplement.box = rect;

    double eps = 1e-2;

    // Truncation assignment based on bbox positions
    if ((ymin < eps) || (ymax >= 1.0 - eps)) {
      obj->camera_supplement.truncated_vertical = 0.5;
    } else {
      obj->camera_supplement.truncated_vertical = 0.0;
    }
    if ((xmin < eps) || (xmax >= 1.0 - eps)) {
      obj->camera_supplement.truncated_horizontal = 0.5;
    } else {
      obj->camera_supplement.truncated_horizontal = 0.0;
    }

    obj->camera_supplement.front_box.xmin *= roi_w;
    obj->camera_supplement.front_box.ymin *= roi_h;
    obj->camera_supplement.front_box.xmax *= roi_w;
    obj->camera_supplement.front_box.ymax *= roi_h;

    obj->camera_supplement.back_box.xmin *= roi_w;
    obj->camera_supplement.back_box.ymin *= roi_h;
    obj->camera_supplement.back_box.xmax *= roi_w;
    obj->camera_supplement.back_box.ymax *= roi_h;

    obj->camera_supplement.front_box.ymin += offset_y;
    obj->camera_supplement.front_box.ymax += offset_y;
    obj->camera_supplement.back_box.ymin += offset_y;
    obj->camera_supplement.back_box.ymax += offset_y;
  }
}

void fill_base(base::ObjectPtr obj, const float *bbox) {
  obj->camera_supplement.box.xmin = bbox[0];
  obj->camera_supplement.box.ymin = bbox[1];
  obj->camera_supplement.box.xmax = bbox[2];
  obj->camera_supplement.box.ymax = bbox[3];
}

void fill_bbox3d(bool with_box3d, base::ObjectPtr obj, const float *bbox) {
  if (with_box3d) {
    obj->camera_supplement.alpha = bbox[0];
    obj->size[2] = bbox[1];
    obj->size[1] = bbox[2];
    obj->size[0] = bbox[3];
  }
}

void fill_frbox(bool with_frbox, base::ObjectPtr obj, const float *bbox) {
  if (with_frbox) {
    obj->camera_supplement.front_box.xmin = bbox[0];
    obj->camera_supplement.front_box.ymin = bbox[1];
    obj->camera_supplement.front_box.xmax = bbox[2];
    obj->camera_supplement.front_box.ymax = bbox[3];

    obj->camera_supplement.back_box.xmin = bbox[4];
    obj->camera_supplement.back_box.ymin = bbox[5];
    obj->camera_supplement.back_box.xmax = bbox[6];
    obj->camera_supplement.back_box.ymax = bbox[7];
  }
}

void fill_lights(bool with_lights, base::ObjectPtr obj, const float *bbox) {
  if (with_lights) {
    obj->car_light.brake_visible = bbox[0];
    obj->car_light.brake_switch_on = bbox[1];
    obj->car_light.left_turn_visible = bbox[2];
    obj->car_light.left_turn_switch_on = bbox[3];
    obj->car_light.right_turn_visible = bbox[4];
    obj->car_light.right_turn_switch_on = bbox[5];
  }
}

void fill_ratios(bool with_ratios, base::ObjectPtr obj, const float *bbox) {
  if (with_ratios) {
    // visible ratios of face a/b/c/d
    obj->camera_supplement.visible_ratios[0] = bbox[0];
    obj->camera_supplement.visible_ratios[1] = bbox[1];
    obj->camera_supplement.visible_ratios[2] = bbox[2];
    obj->camera_supplement.visible_ratios[3] = bbox[3];

    // cut off on width and length (3D)
    obj->camera_supplement.cut_off_ratios[0] = bbox[4];
    obj->camera_supplement.cut_off_ratios[1] = bbox[5];
    // cut off on left and right side (2D)
    obj->camera_supplement.cut_off_ratios[2] = bbox[6];
    obj->camera_supplement.cut_off_ratios[3] = bbox[7];
  }
}

void fill_area_id(bool with_flag, base::ObjectPtr obj, const float *data) {
  if (with_flag) {
    obj->camera_supplement.area_id = static_cast<int>(data[0]);
    // obj->camera_supplement.area_id_prob = data[1];
  }
}

int get_area_id(float visible_ratios[4]) {
  int area_id = 0;
  int max_face = 0;
  for (int i = 1; i < 4; ++i) {
    if (visible_ratios[i] > visible_ratios[max_face]) {
      max_face = i;
    }
  }
  int left_face = (max_face + 1) % 4;
  int right_face = (max_face + 3) % 4;
  const float eps = 1e-3;
  float max_ratio = visible_ratios[max_face];
  float left_ratio = visible_ratios[left_face];
  float right_ratio = visible_ratios[right_face];
  memset(visible_ratios, 0, 4 * sizeof(visible_ratios[0]));
  if (left_ratio < eps && right_ratio < eps) {
    area_id = (max_face * 2 + 1);
    visible_ratios[max_face] = 1.f;
  } else if (left_ratio > right_ratio) {
    area_id = (max_face * 2 + 2);
    auto &&sum_ratio = left_ratio + max_ratio;
    visible_ratios[max_face] = max_ratio / sum_ratio;
    visible_ratios[left_face] = left_ratio / sum_ratio;
  } else {
    area_id = (max_face * 2);
    if (area_id == 0) {
      area_id = 8;
    }
    auto &&sum_ratio = right_ratio + max_ratio;
    visible_ratios[max_face] = max_ratio / sum_ratio;
    visible_ratios[right_face] = right_ratio / sum_ratio;
  }
  return area_id;
}

}  // namespace camera
}  // namespace perception
}  // namespace apollo