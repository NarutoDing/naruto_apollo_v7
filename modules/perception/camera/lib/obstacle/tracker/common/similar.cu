#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2018 The Apollo Authors. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the License);
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an AS IS BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/
#include "modules/perception/camera/lib/obstacle/tracker/common/similar.h"

#include <cblas.h>
#include <memory>

#include "cyber/common/log.h"
#include "modules/perception/camera/common/util.h"
#include "modules/perception/inference/utils/gemm.h"

namespace apollo {
namespace perception {
namespace camera {

bool GPUSimilar::Calc(CameraFrame *frame1, CameraFrame *frame2,
                      base::Blob<float> *sim) {
  int n = static_cast<int>(frame1->detected_objects.size());
  int m = static_cast<int>(frame2->detected_objects.size());
  if ((n && m) == 0) {
    return false;
  }
  sim->Reshape({n, m});

  if (frame1->track_feature_blob == nullptr) {
    AERROR << "No feature blob";
    return false;
  }
  int dim = frame1->track_feature_blob->count(1);
  assert(dim == frame2->track_feature_blob->count(1));

  float *s = sim->mutable_gpu_data();
  float const *feature1 = frame1->track_feature_blob->gpu_data();
  float const *feature2 = frame2->track_feature_blob->gpu_data();
  inference::GPUGemmFloat(CblasNoTrans, CblasTrans, n, m, dim, 1.0, feature1,
                          feature2, 0.0, s);
  return true;
}

}  // namespace camera
}  // namespace perception
}  // namespace apollo
