#include "hip/hip_runtime.h"
/******************************************************************************
COPYRIGHT

All contributions by the University of California:
Copyright (c) 2014-2017 The Regents of the University of California (Regents)
All rights reserved.

All other contributions:
Copyright (c) 2014-2017, the respective contributors
All rights reserved.

Caffe uses a shared copyright model: each contributor holds copyright over
their contributions to Caffe. The project versioning records all such
contribution and copyright details. If a contributor wants to further mark
their specific copyright on a particular contribution, they should indicate
their copyright solely in the commit message of the change when it is
committed.

LICENSE

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.
2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

CONTRIBUTION AGREEMENT

By contributing to the BVLC/caffe repository through pull-request, comment,
or otherwise, the contributor releases their content to the
license and copyright terms herein.
 *****************************************************************************/

/******************************************************************************
 * Copyright 2018 The Apollo Authors. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/
#include <algorithm>
#include <vector>

#include "modules/perception/base/blob.h"
#include "modules/perception/inference/operators/roipooling_layer.h"

namespace apollo {
namespace perception {
namespace inference {
template <typename Dtype>
__global__ void ROIPoolForward(const int nthreads, const Dtype *bottom_data,
                               const bool use_floor, const Dtype spatial_scale,
                               const int channels, const int height,
                               const int width, const int pooled_height,
                               const int pooled_width, const Dtype *bottom_rois,
                               Dtype *top_data, int *argmax_data,
                               const float float_max) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < (nthreads);
       index += blockDim.x * gridDim.x) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    bottom_rois += n * 5;
    int roi_batch_ind = bottom_rois[0];

    int roi_start_w = 0;
    int roi_start_h = 0;
    int roi_end_w = 0;
    int roi_end_h = 0;
    if (use_floor) {
      roi_start_w = floor(bottom_rois[1] * spatial_scale);
      roi_start_h = floor(bottom_rois[2] * spatial_scale);
      roi_end_w = floor(bottom_rois[3] * spatial_scale);
      roi_end_h = floor(bottom_rois[4] * spatial_scale);
    } else {
      roi_start_w = round(bottom_rois[1] * spatial_scale);
      roi_start_h = round(bottom_rois[2] * spatial_scale);
      roi_end_w = round(bottom_rois[3] * spatial_scale);
      roi_end_h = round(bottom_rois[4] * spatial_scale);
    }

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    int roi_height = max(roi_end_h - roi_start_h + 1, 1);
    Dtype bin_size_h =
        static_cast<Dtype>(roi_height) / static_cast<Dtype>(pooled_height);
    Dtype bin_size_w =
        static_cast<Dtype>(roi_width) / static_cast<Dtype>(pooled_width);

    int hstart = static_cast<int>(floor(static_cast<Dtype>(ph) * bin_size_h));
    int wstart = static_cast<int>(floor(static_cast<Dtype>(pw) * bin_size_w));
    int hend = static_cast<int>(ceil(static_cast<Dtype>(ph + 1) * bin_size_h));
    int wend = static_cast<int>(ceil(static_cast<Dtype>(pw + 1) * bin_size_w));

    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart + roi_start_h, 0), height);
    hend = min(max(hend + roi_start_h, 0), height);
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    bool is_empty = (hend <= hstart) || (wend <= wstart);

    // Define an empty pooling region to be zero
    Dtype maxval = is_empty ? 0 : -float_max;
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    int maxidx = -1;
    bottom_data += (roi_batch_ind * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        int bottom_index = h * width + w;
        if (bottom_data[bottom_index] > maxval) {
          maxval = bottom_data[bottom_index];
          maxidx = bottom_index;
        }
      }
    }
    top_data[index] = maxval;
    argmax_data[index] = maxidx;
  }
}
template <typename Dtype>
void ROIPoolingLayer<Dtype>::ForwardGPU(
    const std::vector<std::shared_ptr<base::Blob<Dtype>>> &bottom,
    const std::vector<std::shared_ptr<base::Blob<Dtype>>> &top) {
  auto feat_b = bottom[0];
  auto roi_b = bottom[1];
  channels_ = feat_b->channels();
  height_ = feat_b->height();
  width_ = feat_b->width();

  top[0]->Reshape(roi_b->num(), channels_, pooled_height_, pooled_width_);
  max_idx_.Reshape(roi_b->num(), channels_, pooled_height_, pooled_width_);

  const Dtype *bottom_data = feat_b->gpu_data();
  const Dtype *bottom_rois = roi_b->gpu_data();
  Dtype *top_data = top[0]->mutable_gpu_data();
  int *argmax_data = max_idx_.mutable_gpu_data();
  int count = top[0]->count();
  const int thread_size = 512;
  int block_size = (count + thread_size - 1) / thread_size;
  ROIPoolForward<Dtype><<<block_size, thread_size>>>(
      count, bottom_data, use_floor_, spatial_scale_, channels_, height_,
      width_, pooled_height_, pooled_width_, bottom_rois, top_data, argmax_data,
      float_max_);
}
template void ROIPoolingLayer<double>::ForwardGPU(
    const std::vector<std::shared_ptr<base::Blob<double>>> &bottom,
    const std::vector<std::shared_ptr<base::Blob<double>>> &top);
template void ROIPoolingLayer<float>::ForwardGPU(
    const std::vector<std::shared_ptr<base::Blob<float>>> &bottom,
    const std::vector<std::shared_ptr<base::Blob<float>>> &top);

}  // namespace inference
}  // namespace perception
}  // namespace apollo
