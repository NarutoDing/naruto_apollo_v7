#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2020 The Apollo Authors. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/

#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/sort.h>
#include <thrust/transform.h>

#include "modules/perception/inference/tensorrt/plugins/kernels.h"
#include "modules/perception/inference/tensorrt/plugins/rcnn_proposal_plugin.h"

namespace apollo {
namespace perception {
namespace inference {

// nthreads = num_rois
__global__ void get_rois_nums_kernel(const int nthreads, const float *rois,
                                     int *batch_rois_nums) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    int batch_id = (int)rois[index * 5];
    if (batch_id >= 0) {
      atomicAdd(&batch_rois_nums[batch_id], 1);
    }
  }
}

// bbox_pred dims: [num_rois, box_len, num_class]
// out_bbox_pred dims: [num_rois, num_class, box_len]
__global__ void transpose_bbox_pred_kernel(const int nthreads,
                                           const float *bbox_pred,
                                           const int box_len,
                                           const int num_class,
                                           float *out_bbox_pred) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    int roi_id = index / num_class / box_len;
    int class_id = (index / box_len) % num_class;
    int feature_id = index % box_len;

    int in_index =
        roi_id * box_len * num_class + feature_id * num_class + class_id;
    out_bbox_pred[index] = bbox_pred[in_index];
  }
}

// bbox_pred dims: [num_box, num_class+1, 4],
// scores dims: [num_box, num_class+1],
// out_bbox_pred dims: [num_box, 4]
// out_scores dims: [num_box]
__global__ void get_max_score_kernel(const int nthreads, const float *bbox_pred,
                                     const float *scores, const int num_class,
                                     const float threshold_objectness,
                                     const float *class_thresholds,
                                     float *out_bbox_pred, float *out_scores,
                                     float *out_all_probs, int *filter_count) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index >= nthreads) {
    return;
  }

  int box_id = index;
  if ((1.0f - scores[box_id * (num_class + 1)]) < threshold_objectness) {
    return;
  }

  float score_max = -FLT_MAX;
  int cls_max = -1;
  for (int c = 0; c < num_class; ++c) {
    float score =
        scores[box_id * (num_class + 1) + c + 1] - class_thresholds[c];
    if (score > score_max) {
      score_max = score;
      cls_max = c;
    }
  }
  if (score_max < 0) {
    return;
  } else {
    int counter = atomicAdd(filter_count, 1);
    int box_cls_id = box_id * (num_class + 1) + cls_max + 1;
    for (int i = 0; i < 4; ++i) {
      out_bbox_pred[counter * 4 + i] = bbox_pred[box_cls_id * 4 + i];
    }
    out_scores[counter] = scores[box_cls_id];
    for (int i = 0; i < num_class + 1; ++i) {
      out_all_probs[counter * (num_class + 1) + i] =
          scores[box_id * (num_class + 1) + i];
    }
  }
}

int RCNNProposalPlugin::enqueue(int batchSize, const void *const *inputs,
                                void **outputs, void *workspace,
                                hipStream_t stream) {
  // cls_score_softmax dims: [num_rois, 4, 1, 1]
  const float *cls_score_softmax = reinterpret_cast<const float *>(inputs[0]);
  // bbox_pred dims: [num_rois, 4 * 4 (num_class * box_dim), 1, 1]
  const float *bbox_pred = reinterpret_cast<const float *>(inputs[1]);
  // rois dims: [num_rois, 5, 1, 1]
  const float *rois = reinterpret_cast<const float *>(inputs[2]);
  // im_info dims: [N, 6, 1, 1]
  const float *im_info = reinterpret_cast<const float *>(inputs[3]);
  // output dims: [num_result_box, 9] (axis-1: batch_id, x1, y1, x2, y2,
  //   unknown_score, class1_score, class2_score, class3_score)
  float *result_boxes = reinterpret_cast<float *>(outputs[0]);

  int cls_score_softmax_size = num_rois_ * 4;
  int bbox_pred_size = num_rois_ * 4 * 4;
  int output_size = batchSize * top_n_ * out_channel_;

  // Using thrust::fill might cause crash
  float *init_result_boxes = new float[output_size]();
  std::fill_n(init_result_boxes, output_size, -1.0f);
  BASE_CUDA_CHECK(hipMemcpyAsync(result_boxes, init_result_boxes,
                                  output_size * sizeof(float),
                                  hipMemcpyHostToDevice, stream));

  float *host_im_info = new float[batchSize * 6]();
  BASE_CUDA_CHECK(hipMemcpyAsync(host_im_info, im_info,
                                  batchSize * 6 * sizeof(float),
                                  hipMemcpyDeviceToHost, stream));
  float origin_height = host_im_info[0];
  float origin_width = host_im_info[1];
  float scale = host_im_info[2];

  int nthreads, block_size;

  // TODO(chenjiahao): filter roi that has img_id == -1 at first

  float *host_thresholds = new float[num_class_];
  for (int i = 0; i < num_class_; ++i) {
    host_thresholds[i] = thresholds_[i];
  }
  float *dev_thresholds;
  BASE_CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&dev_thresholds),
                             num_class_ * sizeof(float)));
  BASE_CUDA_CHECK(hipMemcpyAsync(dev_thresholds, host_thresholds,
                                  num_class_ * sizeof(float),
                                  hipMemcpyHostToDevice, stream));

  // Normalize bbox_pred
  float *dev_bbox_mean, *dev_bbox_std;
  float *norm_bbox_pred;
  BASE_CUDA_CHECK(
      hipMalloc(reinterpret_cast<void **>(&dev_bbox_mean), 4 * sizeof(float)));
  BASE_CUDA_CHECK(
      hipMalloc(reinterpret_cast<void **>(&dev_bbox_std), 4 * sizeof(float)));
  BASE_CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&norm_bbox_pred),
                             bbox_pred_size * sizeof(float)));
  BASE_CUDA_CHECK(hipMemcpyAsync(dev_bbox_mean, bbox_mean_, 4 * sizeof(float),
                                  hipMemcpyHostToDevice, stream));
  BASE_CUDA_CHECK(hipMemcpyAsync(dev_bbox_std, bbox_std_, 4 * sizeof(float),
                                  hipMemcpyHostToDevice, stream));
  BASE_CUDA_CHECK(hipMemcpyAsync(norm_bbox_pred, bbox_pred,
                                  bbox_pred_size * sizeof(float),
                                  hipMemcpyDeviceToDevice, stream));
  nthreads = bbox_pred_size;
  block_size = DIVUP(nthreads, thread_size_);
  repeatedly_mul_cuda(block_size, thread_size_, 0, stream, nthreads,
                      norm_bbox_pred, norm_bbox_pred, dev_bbox_std, 4);
  repeatedly_add_cuda(block_size, thread_size_, 0, stream, nthreads,
                      norm_bbox_pred, norm_bbox_pred, dev_bbox_mean, 4);

  // Slice rois
  int slice_axis[4] = {1, 2, 3, 4};
  int *dev_slice_axis;
  float *sliced_rois;
  BASE_CUDA_CHECK(
      hipMalloc(reinterpret_cast<void **>(&dev_slice_axis), 4 * sizeof(int)));
  BASE_CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&sliced_rois),
                             num_rois_ * 4 * sizeof(float)));
  BASE_CUDA_CHECK(hipMemcpyAsync(dev_slice_axis, slice_axis, 4 * sizeof(int),
                                  hipMemcpyHostToDevice, stream));
  nthreads = num_rois_ * 4;
  block_size = DIVUP(nthreads, thread_size_);
  slice2d_cuda(block_size, thread_size_, 0, stream, nthreads, rois, sliced_rois,
               dev_slice_axis, 4, 5);

  // Decode bbox
  float *decoded_bbox_pred;
  BASE_CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&decoded_bbox_pred),
                             bbox_pred_size * sizeof(float)));
  BASE_CUDA_CHECK(hipMemsetAsync(decoded_bbox_pred, 0,
                                  bbox_pred_size * sizeof(float), stream));
  nthreads = num_rois_ * 4;
  block_size = DIVUP(nthreads, thread_size_);
  bbox_transform_inv_cuda(block_size, thread_size_, 0, stream, nthreads,
                          sliced_rois, norm_bbox_pred, num_rois_, 4,
                          decoded_bbox_pred);

  // Refine boxes that are out of map
  if (refine_out_of_map_bbox_) {
    nthreads = bbox_pred_size;
    block_size = DIVUP(nthreads, thread_size_);
    clip_boxes_cuda(block_size, thread_size_, 0, stream, nthreads,
                    decoded_bbox_pred, origin_height, origin_width);
  }

  // Separate data by batch_id
  int *batch_rois_nums = new int[batchSize]();
  int *dev_batch_rois_nums;
  BASE_CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&dev_batch_rois_nums),
                             batchSize * sizeof(int)));
  BASE_CUDA_CHECK(
      hipMemsetAsync(dev_batch_rois_nums, 0, batchSize * sizeof(int), stream));
  nthreads = num_rois_;
  block_size = DIVUP(nthreads, thread_size_);
  get_rois_nums_kernel<<<block_size, thread_size_, 0, stream>>>(
      nthreads, rois, dev_batch_rois_nums);
  BASE_CUDA_CHECK(hipMemcpyAsync(batch_rois_nums, dev_batch_rois_nums,
                                  batchSize * sizeof(int),
                                  hipMemcpyDeviceToHost, stream));

  float *max_bbox, *max_score, *max_all_probs;
  int *max_filtered_count;
  BASE_CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&max_bbox),
                             max_candidate_n_ * 4 * sizeof(float)));
  BASE_CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&max_score),
                             max_candidate_n_ * sizeof(float)));
  BASE_CUDA_CHECK(
      hipMalloc(reinterpret_cast<void **>(&max_all_probs),
                 max_candidate_n_ * (num_class_ + 1) * sizeof(float)));
  BASE_CUDA_CHECK(
      hipMalloc(reinterpret_cast<void **>(&max_filtered_count), sizeof(int)));

  float *filtered_bbox, *filtered_score, *filtered_all_probs;
  int *filtered_count;
  int host_filtered_count;
  BASE_CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&filtered_bbox),
                             max_candidate_n_ * 4 * sizeof(float)));
  BASE_CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&filtered_score),
                             max_candidate_n_ * sizeof(float)));
  BASE_CUDA_CHECK(
      hipMalloc(reinterpret_cast<void **>(&filtered_all_probs),
                 max_candidate_n_ * (num_class_ + 1) * sizeof(float)));
  BASE_CUDA_CHECK(
      hipMalloc(reinterpret_cast<void **>(&filtered_count), sizeof(int)));

  int *sorted_indexes;
  BASE_CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&sorted_indexes),
                             max_candidate_n_ * sizeof(int)));

  float *pre_nms_bbox, *pre_nms_score, *pre_nms_all_probs;
  BASE_CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&pre_nms_bbox),
                             max_candidate_n_ * 4 * sizeof(float)));
  BASE_CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&pre_nms_score),
                             max_candidate_n_ * sizeof(float)));
  BASE_CUDA_CHECK(
      hipMalloc(reinterpret_cast<void **>(&pre_nms_all_probs),
                 max_candidate_n_ * (num_class_ + 1) * sizeof(float)));

  int cur_ptr = 0;
  acc_box_num_ = 0;
  for (int batch_id = 0; batch_id < batchSize; ++batch_id) {
    // TODO(chenjiahao): replace 300 with input dims
    cur_ptr = batch_id * 300;
    BASE_CUDA_CHECK(hipMemsetAsync(
        max_bbox, 0, max_candidate_n_ * 4 * sizeof(float), stream));
    BASE_CUDA_CHECK(hipMemsetAsync(max_score, 0,
                                    max_candidate_n_ * sizeof(float), stream));
    BASE_CUDA_CHECK(hipMemsetAsync(
        max_all_probs, 0, max_candidate_n_ * (num_class_ + 1) * sizeof(float),
        stream));
    BASE_CUDA_CHECK(
        hipMemsetAsync(max_filtered_count, 0, sizeof(int), stream));
    // Get max score among classes and filter with threshold
    nthreads = batch_rois_nums[batch_id];
    block_size = DIVUP(nthreads, thread_size_);
    get_max_score_kernel<<<block_size, thread_size_, 0, stream>>>(
        nthreads, decoded_bbox_pred + size_t(cur_ptr * (num_class_ + 1) * 4),
        cls_score_softmax + size_t(cur_ptr * (num_class_ + 1)), num_class_,
        threshold_objectness_, dev_thresholds, max_bbox, max_score,
        max_all_probs, max_filtered_count);
    int host_max_filtered_count = 0;
    BASE_CUDA_CHECK(hipMemcpyAsync(&host_max_filtered_count,
                                    max_filtered_count, sizeof(int),
                                    hipMemcpyDeviceToHost, stream));
    if (host_max_filtered_count == 0) {
      continue;
    }

    BASE_CUDA_CHECK(hipMemsetAsync(
        filtered_bbox, 0, max_candidate_n_ * 4 * sizeof(float), stream));
    BASE_CUDA_CHECK(hipMemsetAsync(filtered_score, 0,
                                    max_candidate_n_ * sizeof(float), stream));
    BASE_CUDA_CHECK(hipMemsetAsync(
        filtered_all_probs, 0,
        max_candidate_n_ * (num_class_ + 1) * sizeof(float), stream));
    BASE_CUDA_CHECK(hipMemsetAsync(filtered_count, 0, sizeof(int), stream));
    // Filter boxes according to min_size_mode
    nthreads = host_max_filtered_count;
    block_size = DIVUP(nthreads, thread_size_);
    filter_boxes_cuda(block_size, thread_size_, 0, stream, nthreads, max_bbox,
                      max_score, max_all_probs, host_max_filtered_count, 1, 1,
                      num_class_ + 1, 0, 0, min_size_mode_, min_size_h_,
                      min_size_w_, 0.0f, filtered_bbox, filtered_score,
                      filtered_all_probs, filtered_count);

    BASE_CUDA_CHECK(hipMemcpyAsync(&host_filtered_count, filtered_count,
                                    sizeof(int), hipMemcpyDeviceToHost,
                                    stream));
    if (host_filtered_count == 0) {
      continue;
    }

    // Descending sort proposals by score
    thrust::sequence(thrust::device, sorted_indexes,
                     sorted_indexes + host_filtered_count);
    thrust::sort_by_key(thrust::device, filtered_score,
                        filtered_score + size_t(host_filtered_count),
                        sorted_indexes, thrust::greater<float>());

    BASE_CUDA_CHECK(hipMemsetAsync(
        pre_nms_bbox, 0, max_candidate_n_ * 4 * sizeof(float), stream));
    BASE_CUDA_CHECK(hipMemsetAsync(pre_nms_score, 0,
                                    max_candidate_n_ * sizeof(float), stream));
    BASE_CUDA_CHECK(hipMemsetAsync(
        pre_nms_all_probs, 0,
        max_candidate_n_ * (num_class_ + 1) * sizeof(float), stream));
    // Keep max N candidates
    nthreads = std::min(max_candidate_n_, host_filtered_count);
    block_size = DIVUP(nthreads, thread_size_);
    keep_topN_boxes_cuda(
        block_size, thread_size_, 0, stream, nthreads, filtered_bbox,
        filtered_score, filtered_all_probs, sorted_indexes, filtered_count,
        rpn_proposal_output_score_, max_candidate_n_, num_class_ + 1,
        max_candidate_n_, pre_nms_bbox, pre_nms_score, pre_nms_all_probs);

    // NMS
    int cur_filter_count = std::min(host_filtered_count, max_candidate_n_);
    NmsForward(rpn_proposal_output_score_, cur_filter_count, 4, overlap_ratio_,
               max_candidate_n_, top_n_, batch_id, num_class_ + 1, pre_nms_bbox,
               pre_nms_score, pre_nms_all_probs,
               result_boxes + size_t(acc_box_num_ * out_channel_),
               &acc_box_num_, stream);
  }

  // TODO(chenjiahao): rescale bbox

  // Free device memory
  BASE_CUDA_CHECK(hipFree(dev_thresholds));
  BASE_CUDA_CHECK(hipFree(dev_bbox_mean));
  BASE_CUDA_CHECK(hipFree(dev_bbox_std));
  BASE_CUDA_CHECK(hipFree(norm_bbox_pred));
  BASE_CUDA_CHECK(hipFree(dev_slice_axis));
  BASE_CUDA_CHECK(hipFree(sliced_rois));
  BASE_CUDA_CHECK(hipFree(decoded_bbox_pred));
  BASE_CUDA_CHECK(hipFree(dev_batch_rois_nums));
  BASE_CUDA_CHECK(hipFree(max_bbox));
  BASE_CUDA_CHECK(hipFree(max_score));
  BASE_CUDA_CHECK(hipFree(max_filtered_count));
  BASE_CUDA_CHECK(hipFree(filtered_bbox));
  BASE_CUDA_CHECK(hipFree(filtered_score));
  BASE_CUDA_CHECK(hipFree(filtered_count));
  BASE_CUDA_CHECK(hipFree(sorted_indexes));
  BASE_CUDA_CHECK(hipFree(pre_nms_bbox));
  BASE_CUDA_CHECK(hipFree(pre_nms_score));

  // Free host memory
  delete[] init_result_boxes;
  delete[] host_im_info;
  delete[] host_thresholds;
  delete[] batch_rois_nums;
}
}  // namespace inference
}  // namespace perception
}  // namespace apollo