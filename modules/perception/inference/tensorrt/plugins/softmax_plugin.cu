/******************************************************************************
 * Copyright 2018 The Apollo Authors. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/

#include <vector>

#include "modules/perception/inference/tensorrt/plugins/softmax_plugin.h"

namespace apollo {
namespace perception {
namespace inference {

int SoftmaxPlugin::enqueue(int batch_size, const void *const *inputs,
                           void **outputs, void *workspace,
                           hipStream_t stream) {
  const float *in_data = reinterpret_cast<const float *>(inputs[0]);
  float *out_data = reinterpret_cast<float *>(outputs[0]);
  int w = 1;
  int h = inner_num_;
  int c = input_dims_.d[axis_];
  int n = batch_size * outer_num_;
  int w_stride = 1;
  int h_stride = w * w_stride;
  int c_stride = h * h_stride;
  int n_stride = c * c_stride;

  hipdnnSetTensor4dDescriptorEx(input_desc_, HIPDNN_DATA_FLOAT, n, c, h, w,
                               n_stride, c_stride, h_stride, w_stride);
  hipdnnSetTensor4dDescriptorEx(output_desc_, HIPDNN_DATA_FLOAT, n, c, h, w,
                               n_stride, c_stride, h_stride, w_stride);

  float a = 1.0;
  float b = 0.0;
  hipdnnSetStream(cudnn_, stream);
  hipdnnSoftmaxForward(cudnn_, HIPDNN_SOFTMAX_ACCURATE,
                      HIPDNN_SOFTMAX_MODE_CHANNEL, (const void *)(&a),
                      input_desc_, in_data, (const void *)(&b), output_desc_,
                      out_data);

  return 1;
}
}  // namespace inference
}  // namespace perception
}  // namespace apollo
