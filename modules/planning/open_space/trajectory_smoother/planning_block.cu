#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2019 The Apollo Authors. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/

#include <iostream>

#include "planning_block.h"

namespace apollo {
namespace planning {
bool InitialCuda() {
  int dev = 0;
  hipDeviceProp_t deviceProp;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, dev));
  CUDA_CHECK(hipSetDevice(dev));
  return true;
}

__global__ void fill_lower_left_gpu(int *iRow, int *jCol, unsigned int *rind_L,
                                    unsigned int *cind_L, const int nnz_L) {
  int i = threadIdx.x;

  if (i < nnz_L) {
    iRow[i] = rind_L[i];
    jCol[i] = cind_L[i];
  }
}

template <typename T>
__global__ void data_transfer_gpu(T *dst, const T *src, const int size) {
  int i = threadIdx.x;

  if (i < size) {
    dst[i] = src[i];
  }
}

bool fill_lower_left(int *iRow, int *jCol, unsigned int *rind_L,
                     unsigned int *cind_L, const int nnz_L) {
  if (!InitialCuda()) return false;
  int *d_iRow, *d_jCol;
  unsigned int *d_rind_L, *d_cind_L;

  unsigned int nBytes = nnz_L * sizeof(int);
  unsigned int nUBytes = nnz_L * sizeof(unsigned int);
  hipMalloc((void **)&d_iRow, nBytes);
  hipMalloc((void **)&d_jCol, nBytes);
  hipMalloc((void **)&d_rind_L, nUBytes);
  hipMalloc((void **)&d_cind_L, nUBytes);

  hipMemcpy(d_iRow, iRow, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_jCol, jCol, nBytes, hipMemcpyHostToDevice);

  dim3 block(BLOCK_1);
  dim3 grid((nnz_L + block.x - 1) / block.x);

  fill_lower_left_gpu<<<grid, block>>>(d_iRow, d_jCol, d_rind_L, d_cind_L,
                                       nnz_L);
  hipDeviceSynchronize();

  hipMemcpy(rind_L, d_rind_L, nUBytes, hipMemcpyDeviceToHost);
  hipMemcpy(cind_L, d_cind_L, nUBytes, hipMemcpyDeviceToHost);

  hipFree(d_iRow);
  hipFree(d_jCol);
  hipFree(d_rind_L);
  hipFree(d_cind_L);
  hipDeviceReset();
  return true;
}

template <typename T>
bool data_transfer(T *dst, const T *src, const int size) {
  if (!InitialCuda()) return false;
  T *d_dst, *d_src;
  size_t nBytes = size * sizeof(T);
  hipMalloc((void **)&d_dst, nBytes);
  hipMalloc((void **)&d_src, nBytes);
  hipMemcpy(d_src, src, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_dst, dst, nBytes, hipMemcpyHostToDevice);

  dim3 block(BLOCK_1);
  dim3 grid((size + block.x - 1) / block.x);

  data_transfer_gpu<<<grid, block>>>(dst, src, size);
  hipDeviceSynchronize();

  hipMemcpy(dst, d_dst, nBytes, hipMemcpyDeviceToHost);

  hipFree(d_dst);
  hipFree(d_src);
  hipDeviceReset();
  return true;
}

DATA_TRANSFER_INST(int);
DATA_TRANSFER_INST(double);
DATA_TRANSFER_INST(float);

}  // namespace planning
}  // namespace apollo
